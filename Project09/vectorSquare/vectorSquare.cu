/**
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 * Extended for use in CS 374 at Calvin College by Joel C. Adams.
 */

/**
 * Vector square of A = C.
 *
 * This sample is a very basic sample that implements element by element
 * vector square. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some squares like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

// For the runtimes using OpenMP
#include <omp.h>

/**
 * CUDA Kernel Device code
 *
 * Computes the vectorSquare A * A = C. 
 * The 3 vectors have the same number of elements numElements.
 */
__global__
void vectorSquare(const float *A, float *C, unsigned long numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] * A[i];
    }
}

void checkErr(hipError_t err, const char* msg) 
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "%s (error code %d: '%s')!\n", msg, err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/**
 * Host main routine
 */
int main(int argc, char** argv)
{
    // Timing varibles
    double startCuda, finalCuda;
    double startSequential, finalSequential;
    double startSum, finalSum;
    double startHostToDev, finalHostToDev;
    double startDevToHost, finalDevToHost;

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    unsigned long numElements = 50000;
    if (argc == 2) {
      numElements = strtoul( argv[1] , 0, 10 );
    }
    size_t size = numElements * sizeof(float);
    printf("[Vector Square of %lu elements]\n", numElements);

    // Allocate the host input vectors A
    float * h_A = (float *)malloc(size);

    // Allocate the host output vector C
    float * h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
    }

    // 1a. Allocate the device input vectors A
    float * d_A = NULL;
    err = hipMalloc((void **)&d_A, size);
    checkErr(err, "Failed to allocate device vector A");

    // 1.b. Allocate the device output vector C
    float * d_C = NULL;
    err = hipMalloc((void **)&d_C, size);
    checkErr(err, "Failed to allocate device vector C");

    // 2. Copy the host input vector A in host memory 
    //     to the device input vectors in device memory
    startCuda = omp_get_wtime();                                            //start Cuda total time
    startHostToDev = omp_get_wtime();                                       //start host to dev time
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    checkErr(err, "Failed to copy device vector A from host to device");
    finalHostToDev = omp_get_wtime() - startHostToDev;                      //stop host to dev time

    // 3. Launch the Vector square CUDA Kernel
    startSum = omp_get_wtime();                                             //start computing of sum Time
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorSquare<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_C, numElements);
    err = hipGetLastError();
    checkErr(err, "Failed to launch vectorSquare kernel");
    finalSum = omp_get_wtime() - startSum;                                  //stop computing of sum time


    // 4. Copy the device result vector in device memory
    //     to the host result vector in host memory.
    startDevToHost = omp_get_wtime();                                       //start Dev to Host time
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    checkErr(err, "Failed to copy vector C from device to host");
    finalDevToHost = omp_get_wtime() - startDevToHost;                      //stop Dev to Host
    finalCuda = omp_get_wtime() - startCuda;                                //stop Cuda time

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs( h_A[i] * h_A[i] - h_C[i] ) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("CUDA test PASSED\n");
//    printf("CUDA time: %lf\n", stopTime-startTime); 
    printf("Host to dev time: %lf\n", finalHostToDev);
    printf("Computation Sum time: %lf\n", finalSum);
    printf("Dev to Host time: %lf\n", finalDevToHost);
    printf("Cuda runtime: %lf\n", finalCuda);

    // Free device global memory
    err = hipFree(d_A);
    checkErr(err, "Failed to free device vector A");

    err = hipFree(d_C);
    checkErr(err, "Failed to free device vector C");

    // repeat the computation sequentially
    startSequential = omp_get_wtime();                              //start Sequential time
    for (int i = 0; i < numElements; ++i)
    {
       h_C[i] = h_A[i] * h_A[i];
    }
    finalSequential = omp_get_wtime() - startSequential;            //stop sequential time
    printf("Sequential time: %lf\n", finalSequential);


    // verify again
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs( h_A[i] * h_A[i] - h_C[i] ) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("\nNormal test PASSED\n");
//    printf("Normal time: %lf\n", stopTime-startTime); 
    
    // Free host memory
    free(h_A);
    free(h_C);

    // Reset the device and exit
    err = hipDeviceReset();
    checkErr(err, "Unable to reset device");

    printf("Done\n");
    return 0;
}

